#include "element.h"
#include "../culib/gpuVector.cuh"
#include ""
using namespace gv;

__constant__ float* coef_g[576 * 9];

void uploadcoef(float* ptr)
{
	hipMemcpyToSymbol(HIP_SYMBOL(coef_g), ptr, 576 * 9);
	cuda_error_check;
}

void spinodal::init_gpu()
{
	hipMalloc(&gbuf.temp, 9 * nel * sizeof(float));
	hipMemcpy(gbuf.temp, temp, 9 * nel * sizeof(float), hipMemcpyHostToDevice);
	uploadcoef(coef.data());
	cuda_error_check;
}

void spinodal::free_gpu()
{
	hipFree(gbuf.temp);
	cuda_error_check;
}

void spinodal::value_gpu(const spinodal& inst)
{
	hipFree(gbuf.temp);
	hipMalloc(&gbuf.temp, 9 * inst.nel * sizeof(float));
	hipMemcpy(gbuf.temp, inst.gbuf.temp, 9 * inst.nel * sizeof(float), hipMemcpyDeviceToDevice);
	cuda_error_check;
}