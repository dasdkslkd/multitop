#include "element.h"
#include "hip/hip_runtime.h"
#include ""
//#include "../culib/gMat.cuh"
//#include "../culib/cudaCommon.cuh"
//#include "../culib/gpuVector.cuh"
//using namespace gv;

__constant__ double* coef_g[576 * 9];

void uploadcoef(double* ptr)
{
	hipMemcpyToSymbol(HIP_SYMBOL(coef_g), ptr, 576 * 9);
	cuda_error_check;
}

void spinodal::init_gpu()
{
	//hipMalloc(&gbuf.temp, 9 * nel * sizeof(double));
	//hipMemcpy(gbuf.temp, temp, 9 * nel * sizeof(double), hipMemcpyHostToDevice);
	uploadcoef(coef.data());
	gbuf.temp.set_from_host(temp, 9, nel);
	cuda_error_check;
}

void spinodal::free_gpu()
{
	//hipFree(gbuf.temp);

	cuda_error_check;
}

void spinodal::value_gpu(const spinodal& inst)
{
	//hipFree(gbuf.temp);
	//hipMalloc(&gbuf.temp, 9 * inst.nel * sizeof(double));
	//hipMemcpy(gbuf.temp, inst.gbuf.temp, 9 * inst.nel * sizeof(double), hipMemcpyDeviceToDevice);
	gbuf.temp.clear();
	gbuf.temp = inst.gbuf.temp;
	
	cuda_error_check;
}